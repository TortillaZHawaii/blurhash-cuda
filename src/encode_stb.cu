#include "../include/encode.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "../include/stb/stb_image.h"

#include <stdio.h>

const char *blurHashForFile(int xComponents, int yComponents,const char *filename);

int main(int argc, const char **argv) 
{
	if(argc != 4 && argc != 5)
    {
		fprintf(stderr, "Usage: %s x_components y_components imagefile [csvfile]\n", argv[0]);
		return 1;
	}

	int xComponents = atoi(argv[1]);
	int yComponents = atoi(argv[2]);
	
    if(xComponents < 1 || xComponents > 8 || yComponents < 1 || yComponents > 8)
    {
		fprintf(stderr, "Component counts must be between 1 and 8.\n");
		return 1;
	}

	const char *hash = blurHashForFile(xComponents, yComponents, argv[3]);
	
    if(!hash) 
    {
		fprintf(stderr, "Failed to load image file \"%s\".\n", argv[3]);
		return 1;
	}

	printf("%s\n", hash);

	return 0;
}

const char *blurHashForFile(int xComponents, int yComponents,const char *filename)
{
	int width, height, channels;
	unsigned char *data = stbi_load(filename, &width, &height, &channels, 3);
	if(!data) return NULL;

	const char *hash = blurHashForPixels(xComponents, yComponents, width, height, data, width * 3);

	stbi_image_free(data);

	return hash;
}
