#include "hip/hip_runtime.h"
#include "../include/encode.cuh"
#include "../include/csv/csv_logger.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "../include/stb/stb_image.h"

#include <stdio.h>
#include <time.h>

const char *blurHashForFile(int xComponents, int yComponents,
	const char *filename, const char *csvFilename);

int main(int argc, const char **argv) 
{
	if(argc != 4 && argc != 5)
    {
		fprintf(stderr, "Usage: %s x_components y_components imagefile [csvfile]\n", argv[0]);
		return 1;
	}

	int xComponents = atoi(argv[1]);
	int yComponents = atoi(argv[2]);
	
    if(xComponents < 1 || xComponents > 8 || yComponents < 1 || yComponents > 8)
    {
		fprintf(stderr, "Component counts must be between 1 and 8.\n");
		return 1;
	}

	const char *csvFilename = argc == 5 ? argv[4] : NULL;

	const char *hash = blurHashForFile(xComponents, yComponents, argv[3], csvFilename);
	
    if(!hash) 
    {
		fprintf(stderr, "Failed to load image file \"%s\".\n", argv[3]);
		return 1;
	}

	printf("%s\n", hash);

	return 0;
}

const char *blurHashForFile(int xComponents, int yComponents, 
	const char *filename, const char *csvFilename)
{
	int width, height, channels;
	unsigned char *data = stbi_load(filename, &width, &height, &channels, 3);
	if(!data) return NULL;

	clock_t startEncode = clock();

	const char *hash = blurHashForPixels(xComponents, yComponents, width, height, data, width * 3);

	clock_t endEncode = clock();

	if(csvFilename != nullptr)
	{
		int msec = (endEncode - startEncode) * 1000 / CLOCKS_PER_SEC;
		csvAppendEncoderLogs(csvFilename, filename, width, height,
		 	hash, xComponents, yComponents, "Pure C", msec);
	}


	stbi_image_free(data);

	return hash;
}
