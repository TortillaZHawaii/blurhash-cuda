#include "../include/decode.cuh"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../include/stb/stb_writer.h"

int main(int argc, char **argv)
{
	if(argc < 5) 
	{
		fprintf(stderr, "Usage: %s hash width height output_file [punch]\n", argv[0]);
		return 1;
	}

	int width, height, punch = 1;
	char * hash = argv[1];
	width = atoi(argv[2]);
	height = atoi(argv[3]);
	char * output_file = argv[4];

	const int nChannels = 4;

	if(argc == 6)
		punch = atoi(argv[5]);

	uint8_t * bytes = decode(hash, width, height, punch, nChannels);

	if (!bytes)
	{
		fprintf(stderr, "%s is not a valid blurhash, decoding failed.\n", hash);
		return 1;
	}

	if (stbi_write_png(output_file, width, height, nChannels, bytes, nChannels * width) == 0)
	{
		fprintf(stderr, "Failed to write PNG file %s\n", output_file);
		return 1;
	}

	freePixelArray(bytes);

	fprintf(stdout, "Decoded blurhash successfully, wrote PNG file %s\n", output_file);
	return 0;
}
